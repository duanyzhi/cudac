#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <math.h> 
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum) {
    if (WarpSize >= 32)sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16)sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8)sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4)sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2)sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

// if N <= 16
template <
    const int ROW_PER_WARP
    > 
__global__ void Sgemv_v2( 
    float * __restrict__ A,
    float * __restrict__ x,
    float * __restrict__ y, 
    const int M,
    const int N) {
    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int warp_size=32;
    int laneId= tx % warp_size;
    int current_warp_row = (blockDim.y * bx + ty) * ROW_PER_WARP;
    const int kWarp_size = warp_size / ROW_PER_WARP;
    int kLaneId = laneId % kWarp_size;
    int current_thread_row = current_warp_row + laneId / kWarp_size;

    if(current_thread_row < M){
        float res=0;
        int current_col = kLaneId;
        res += A[current_thread_row * N + current_col] * x[current_col];
        res = warpReduceSum<kWarp_size>(res);
        if(kLaneId==0) y[current_thread_row]=res;
    }
}

at::Tensor gemv_forward(at::Tensor a, at::Tensor b) {
    hipStream_t stream = c10::cuda::getCurrentCUDAStream();

    const int M = a.size(0);
    const int K = a.size(1);
    const int N = b.size(0);

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("gemm", &gemm_forward, "Gemm forward (CUDA)",
          py::arg("a"), py::arg("b"));
}