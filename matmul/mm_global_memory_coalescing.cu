#include "hip/hip_runtime.h"
#include "fun.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void mm_cuda_memory_coalescing(float* A, float* B, float* C, int M, int N, int K) {
  // int row = blockDim.x * blockIdx.x + threadIdx.x;
  // int col = blockDim.y * blockIdx.y + threadIdx.y;

  const int row = blockIdx.x * BLOCK_SIZE + (threadIdx.x / BLOCK_SIZE);
  const int col = blockIdx.y * BLOCK_SIZE + (threadIdx.x % BLOCK_SIZE);

  if (row >= M) return;
  if (col >= K) return;

  // printf("bidx %d, tx %d, bixy %d, ty %d\n", blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);
  // printf("id x y %d, %d\n", row, col);
  float sum_c = 0;
  for (int i = 0; i < N; ++i) {
    sum_c += A[row * N + i] *
             B[col + i * K];
  }
  C[row * K + col] = sum_c;
}


void run_mm_cuda_memory_coalescing(float* hA, float* hB, float* hC, int M, int N, int K) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  size_t size_A = M * N * sizeof(float);
  size_t size_B = N * K * sizeof(float);
  size_t size_C = M * K * sizeof(float);

  float* dA;
  float* dB;
  float* dC;
  hipMalloc((void**)&dA, size_A);
  hipMalloc((void**)&dB, size_B);
  hipMalloc((void**)&dC, size_C);

  // hipMemcpy
  hipMemcpy((void*)dA, (void*)hA, size_A, hipMemcpyHostToDevice);
  hipMemcpy((void*)dB, (void*)hB, size_B, hipMemcpyHostToDevice);

  hipEventRecord(start, 0);

  dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE);
  dim3 dimGrid((M + BLOCK_SIZE - 1) / BLOCK_SIZE,
               (K + BLOCK_SIZE - 1) / BLOCK_SIZE);
  mm_cuda_memory_coalescing<<<dimGrid, dimBlock>>>(dA, dB, dC, M, N, K);

  // wait for synchronize
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  // copy cuda to cpu
  hipMemcpy((void*)hC, (void*)dC, size_C, hipMemcpyDeviceToHost);
  
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("cuda mm gpu memory coalescing time for [%d, %d, %d] is %f ms.\n", M, N, K, milliseconds);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}
