#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "fun.h"

// cuBLAS SGEMM 
void run_mm_cuda_cublas(float* hA, float* hB, float* hC, int M, int K, int N) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipError_t cudaStat;  // hipMalloc status
  hipblasStatus_t stat;   // cuBLAS functions status
  hipblasHandle_t handle; // cuBLAS context

  size_t size_A = M * K * sizeof(float);
  size_t size_B = K * N * sizeof(float);
  size_t size_C = M * N * sizeof(float);

  float* dA;
  float* dB;
  float* dC;
  hipMalloc((void**)&dA, size_A);
  hipMalloc((void**)&dB, size_B);
  hipMalloc((void**)&dC, size_C);

  // run kernel
  stat = hipblasCreate(&handle); // initialize CUBLAS context
  float alpha = 1.0f;
  float beta = 0.0f;

  // hipMemcpy
  hipMemcpy((void*)dA, (void*)hA, size_A, hipMemcpyHostToDevice);
  hipMemcpy((void*)dB, (void*)hB, size_B, hipMemcpyHostToDevice);

  // m -> M, n -> N, k -> K
  // ref: https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-gemmex
  // waraup
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB, N,
                     dA, K, &beta, dC, N);


  hipEventRecord(start, 0);
  for (int i = 0; i < 10; ++i) {
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dB, N,
                     dA, K, &beta, dC, N);
  }


  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  // copy cuda to cpu
  hipMemcpy((void*)hC, (void*)dC, size_C, hipMemcpyDeviceToHost);
  
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  hipblasDestroy(handle); // destroy CUBLAS context

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("cuda mm cublas time for [%d, %d, %d] is %f ms.\n", M, K, N, milliseconds / 10);
}
