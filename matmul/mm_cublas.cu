#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "fun.h"

// cuBLAS SGEMM 
void run_mm_cuda_cublas(float* hA, float* hB, float* hC, int M, int N, int K) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  hipError_t cudaStat;  // hipMalloc status
  hipblasStatus_t stat;   // cuBLAS functions status
  hipblasHandle_t handle; // cuBLAS context

  size_t size_A = M * N * sizeof(float);
  size_t size_B = N * K * sizeof(float);
  size_t size_C = M * K * sizeof(float);

  float* dA;
  float* dB;
  float* dC;
  hipMalloc((void**)&dA, size_A);
  hipMalloc((void**)&dB, size_B);
  hipMalloc((void**)&dC, size_C);

  // run kernel
  stat = hipblasCreate(&handle); // initialize CUBLAS context
  float alpha = 1.0f;
  float beta = 0.0f;

  // hipMemcpy
  hipMemcpy((void*)dA, (void*)hA, size_A, hipMemcpyHostToDevice);
  hipMemcpy((void*)dB, (void*)hB, size_B, hipMemcpyHostToDevice);

  hipEventRecord(start, 0);
  // m -> M, n -> K, k -> N
  // ref: https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-gemmex
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, M, N, &alpha, dB, K,
                     dA, N, &beta, dC, K);

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  // copy cuda to cpu
  hipMemcpy((void*)hC, (void*)dC, size_C, hipMemcpyDeviceToHost);
  
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  hipblasDestroy(handle); // destroy CUBLAS context

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("cuda mm cublas time for [%d, %d, %d] is %f ms.\n", M, N, K, milliseconds);
}
