#include "hip/hip_runtime.h"
#include "fun.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void mm_cuda_naive(float* A, float* B, float* C, int M, int N, int K) {
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  int col = blockDim.y * blockIdx.y + threadIdx.y;

  if (row >= M) return;
  if (col >= K) return;

  float sum_c = 0;
  for (int i = 0; i < N; ++i) {
    sum_c += A[row * N + i] *
             B[col + i * K];
  }
  C[row * K + col] = sum_c;
}


void run_mm_cuda_naive(float* hA, float* hB, float* hC, int M, int N, int K) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  size_t size_A = M * N * sizeof(float);
  size_t size_B = N * K * sizeof(float);
  size_t size_C = M * K * sizeof(float);

  float* dA;
  float* dB;
  float* dC;
  hipMalloc((void**)&dA, size_A);
  hipMalloc((void**)&dB, size_B);
  hipMalloc((void**)&dC, size_C);

  // hipMemcpy
  hipMemcpy((void*)dA, (void*)hA, size_A, hipMemcpyHostToDevice);
  hipMemcpy((void*)dB, (void*)hB, size_B, hipMemcpyHostToDevice);


  hipEventRecord(start, 0);
  //// run kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((M + dimBlock.x - 1) / dimBlock.x,
               (K + dimBlock.y - 1) / dimBlock.y);

  mm_cuda_naive<<<dimGrid, dimBlock>>>(dA, dB, dC, M, N, K);

  // wait for synchronize
  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize( stop );

  // copy cuda to cpu
  hipMemcpy((void*)hC, (void*)dC, size_C, hipMemcpyDeviceToHost);
  
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("cuda mm naive time for [%d, %d, %d] is %f ms.\n", M, N, K, milliseconds);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}
