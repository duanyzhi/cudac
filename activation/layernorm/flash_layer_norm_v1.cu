#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/AccumulateType.h>
#include <iostream>
#include <hip/hip_fp16.h>
#include <ATen/cuda/HIPContext.h>
#include "pybind.h"

const unsigned int WARP_REDUCE_MASK = 0xffffffff;
#define MAX_THREADS 1024

__device__ void warp_sum(double* __restrict__ val) {
  *val += __shfl_xor_sync(WARP_REDUCE_MASK, *val, 16, 32);
  *val += __shfl_xor_sync(WARP_REDUCE_MASK, *val, 8, 32);
  *val += __shfl_xor_sync(WARP_REDUCE_MASK, *val, 4, 32);
  *val += __shfl_xor_sync(WARP_REDUCE_MASK, *val, 2, 32);
  *val += __shfl_xor_sync(WARP_REDUCE_MASK, *val, 1, 32);
}

__global__ void layernorm(const float4* __restrict__ x, const float4* __restrict__ weight,
    const float4* __restrict__ bias, float eps, float4* __restrict__ output, int hidden_dim) {
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    const float4* x_block = x + blockIdx.x * hidden_dim;

    double sum_x = 0;
    double sum_xx = 0;
    #pragma unroll
    for (uint id = tid; id < hidden_dim; id+=blockDim.x) {
      float4 val = x_block[id];
      sum_x += val.x + val.y + val.z + val.w;
      sum_xx +=
          val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
    }

    // __syncthreads();
    // warp sum to get 1024 / 32 sum
    warp_sum(&sum_x);
    warp_sum(&sum_xx);

    __syncthreads();

    int lane_id = threadIdx.x & 0x1f;
    int wid = threadIdx.x >> 5;

    static __shared__ double shared_mean[32];  // one warp for sum
    static __shared__ double shared_var[32];  // one warp for sum
    if (lane_id == 0) {  // push all lane_id 0 to shared
      shared_mean[wid] = sum_x;
      shared_var[wid] = sum_xx;
      // printf("lane id %d wid  %d mean %f", lane_id, wid, mean);
    }

    __syncthreads();
    if (threadIdx.x < (blockDim.x >> 5)) {  // < 32
      sum_x = shared_mean[lane_id];  // get mean from shared memory
      sum_xx = shared_var[lane_id];  // get mean from shared memory
    } else {
      sum_x = 0.0f;  // only keey warp data
      sum_xx = 0.0f;  // only keey warp data
    }
    warp_sum(&sum_x);
    warp_sum(&sum_xx);
    __syncthreads();

    __shared__ double reduce_mean, reduce_var;
    if (tid == 0) {
       reduce_mean = __ddiv_rn(sum_x, hidden_dim * 4.0f);
       reduce_var = __ddiv_rn(sum_xx, hidden_dim * 4.0f) - __dmul_rn(reduce_mean, reduce_mean) + eps;
       reduce_var = rsqrtf(reduce_var);
    }
    __syncthreads(); // wait reduce_mean and var

    // compute results
    float4 *output_block = output + blockIdx.x * hidden_dim;
    // #pragma unroll
    // for (uint id = tid; id < hidden_dim; id+=blockDim.x) {
    //   float alpha  = __ldg(weight + tid);
    //   float beta  = __ldg(bias + tid);
    //   output_block[id] = __ddiv_rn((x_block[id] - reduce_mean), reduce_var) * alpha + beta;
    // }

    for (uint idx = tid; idx < hidden_dim; idx += blockDim.x) {
      float4 vscale = __ldg(reinterpret_cast<const float4 *>(weight) + idx);
      float4 vbias = __ldg(reinterpret_cast<const float4 *>(bias) + idx);
      float4 val = x_block[idx];
      val.x = (val.x - reduce_mean) * reduce_var * vscale.x + vbias.x;
      val.y = (val.y - reduce_mean) * reduce_var * vscale.y + vbias.y;
      val.z = (val.z - reduce_mean) * reduce_var * vscale.z + vbias.z;
      val.w = (val.w - reduce_mean) * reduce_var * vscale.w + vbias.w;
      output_block[idx] = val;
    }
}

at::Tensor flash_layernorm(at::Tensor x, at::Tensor weight,
    at::Tensor bias, double eps) {

  auto output = torch::empty_like(x);
  // std::cout << "input tensor scaler type " << x.scalar_type() << "\n";
  // std::cout << "weight tensor scaler type " << weight.scalar_type() << "\n";

  const int batch_size = x.size(0);
  int hidden_dim = 1;

  auto sizes = x.sizes();
  for (int i = 1; i < sizes.size(); ++i) {
      hidden_dim = hidden_dim * sizes[i];
  }

  // for (int i = 1; i < x.size().size(); ++i) {
  //    hidden_dim *= x.size(i);
  // }
  // // const int hidden_dim = x.size(1);
  // std::cout << "hidden_dim " << hidden_dim << "\n";

  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);
  // std::cout << "grid dim " << batch_size << " block dim " << block_dim.x << ", " << block_dim.y << "\n";

  const float4* x_f4 = reinterpret_cast<const float4 *>(x.data_ptr()); 
  const float4* w_f4 = reinterpret_cast<const float4 *>(weight.data_ptr()); 
  const float4* b_f4 = reinterpret_cast<const float4 *>(bias.data_ptr()); 
  float4* o_f4 = reinterpret_cast<float4 *>(output.data_ptr()); 
  hidden_dim = hidden_dim / 4;

  layernorm<<<grid_dim, block_dim>>>(
      x_f4, w_f4, b_f4,
      static_cast<float>(eps),
      o_f4, hidden_dim);
  return output;
}
